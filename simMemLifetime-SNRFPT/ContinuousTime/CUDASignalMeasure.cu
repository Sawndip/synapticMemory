#include "hip/hip_runtime.h"


#include  "../stdafx.h"
//#include "scalarProd_kernel.cu"

//Prototype
__global__ void scalarSigProdGPU(
    float *d_C,
    int *d_A,
    int *d_W,
    int vectorN,
    int elementN
);

// Enable this for error checking
#define CUDA_CHECK_ERROR

#define CudaSafeCall( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()        __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_CHECK_ERROR

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);

    do
    {
        if ( hipSuccess != err )
        {
            fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );

#pragma warning( pop )

#endif  // CUDA_CHECK_ERROR

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_CHECK_ERROR

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);

    do
    {
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err )
        {
            fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }

        // More careful checking. However, this will affect performance.
        // Comment if not needed.
        err = hipDeviceSynchronize();
        if( hipSuccess != err )
        {
            fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
                     file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );

#pragma warning( pop )

#endif // CUDA_CHECK_ERROR

    return;
}

#ifndef _SIGNAL_KERNEL_H_
#define _SIGNAL_KERNEL_H_


#define MAX_BLOCK_DIM_SIZE 65535

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};


// Device code
__global__ void VecMult(int* X,int* W, int* C, unsigned int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = X[i]*W[i];
}


template <class T>
__global__ void
reduce3(T *g_idata, T *g_odata, uint n)
{
    T *sdata = SharedMemory<T>();
   // T *sdataW = SharedMemory<T>();
    
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : 0;
    if (i + blockDim.x < n) 
        mySum += g_idata[i+blockDim.x];  

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s) 
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


void initCUDADeviceMem(void*& d_W,void*& d_X,void*& d_C,void*& d_odata,float*& h_odata,unsigned int _uiSynCount,uint _uiTrackCount)
{
	 uint size = _uiSynCount*sizeof(int); //Memory Size for Input Vector
	 int threadsPerBlock = 512;
	 int N = _uiSynCount;
	 //int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	    
	 CudaSafeCall( hipMalloc((void**)&d_W, size) ); //Weight V
	 CudaSafeCall( hipMalloc((void**)&d_X,size*_uiTrackCount) ); //Input V
	 cout << "P:" << d_X << endl;
	 CudaSafeCall( hipMalloc((void**)&d_C, _uiTrackCount*sizeof(float)) ); //Result Mux Vector Memory Space
	 //CudaSafeCall( hipMalloc((void**)&d_odata, blocksPerGrid*sizeof(float)) ); //Reduction Results Per block

	 h_odata = new float[_uiTrackCount]; //The Output of each GPU Block - Copied to the Host
	 memset(h_odata,1,sizeof(float)*_uiTrackCount);
}


void cleapUpCUDADeviceMem(void*& d_W,void*& d_X,void*& d_C,void*& d_odata,float*& h_odata,uint _uiSynCount)
{

		 delete [] h_odata;
	    //Clean Up Device
		cout << "P:" << d_X  << endl;
		CudaSafeCall( hipFree(d_X) );
		CudaSafeCall( hipFree(d_C) );
		//CudaSafeCall( hipFree(d_odata) );
		CudaSafeCall( hipFree(d_W) );
	
}

void transferVectorsToDevice(int iNoTrackedPats,int* h_W ,t_inVal* h_X, uint _uiSynCount,void*& d_W,void*& d_X)
{
	uint size = _uiSynCount*sizeof(t_inVal);
    CudaSafeCall( hipMemcpy((void*)d_W, (const void*)h_W, size, hipMemcpyHostToDevice) ); //Weight
   	CudaSafeCall( hipMemcpy((void*)d_X, (const void*)h_X, size*iNoTrackedPats, hipMemcpyHostToDevice) ); //Input Patterns
	
}

//Get Perceptron Signal But do not Use Synapses Pointed by track group
// iTrackedIndex : Give the index of the currently tested Tracked pattern from the list of tracked Patts - Copy Weight Vector only the 1st time optimization
//Return The number of Tracked Patterns That can be tested now given how many patterns have been stored
int testCUDAPRecallOfX(float* h_sigdata,t_inVal* W ,t_inVal** X,t_inVal* tX, uint _uiSynCount,void* d_W,void* d_X,void* d_C,t_patt_trackedtbl& vTrackedIndex,uint _uiPattsStoredSoFar)
{
	//Test Recall of _iStoredPatIndex
	//cout << "Recall index: " << _iStoredPatIndex << " Output should be :" << X[_iStoredPatIndex][_uiSynCount-1];
	uint N = _uiSynCount;
	//uint size = N*sizeof(int);
    //int threadsPerBlock = 512;
   // int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	int i=0;
	//assert(uiNoOfPatternsStoredInTrial > uiInitPatterns);
	t_patt_trackedtbl::iterator itTracked;
	for (itTracked = vTrackedIndex.begin();itTracked!=vTrackedIndex.end();++itTracked) //Copy Tracked Patterns that have Occured Up To now
	{
		if (itTracked->first < (_uiPattsStoredSoFar)) //Only If Tracked pattern Has been created - Note We are Comparing Index To Patt count
			memcpy(&tX[i*_uiSynCount],X[itTracked->first],sizeof(t_inVal)*_uiSynCount); //Join All tracked Patterns into one vector
		i++;
	}
	int iTrackedCount = i;
	//cout << W[0] << "X:" << X[0] << " N:" << _uiSynCount << " TP:"<<iTrackedCount << " dW"<<d_W << " dX"<<d_X << endl;
	// Allocate vectors in device memory happens once at beginning
	// Copy vectors from host memory to device memory
    //Only Copy One Weight Vector Which is used Against all Tracked Patterns
    //CudaSafeCall( hipMemcpy((void*)d_W, (const void*)W, size, hipMemcpyHostToDevice) ); //Weight
   //	CudaSafeCall( hipMemcpy((void*)d_X, (const void*)X, size*iNoTrackedPats, hipMemcpyHostToDevice) ); //Input Patterns
     transferVectorsToDevice(iTrackedCount,W,tX,_uiSynCount,d_W,d_X);
	
//	  CudaSafeCall( hipMemcpy((void*)d_W, (const void*)W, size, hipMemcpyHostToDevice) ); //Weight
//	  CudaSafeCall( hipMemcpy((void*)d_X, (const void*)X, size*iNoTrackedPats, hipMemcpyHostToDevice) ); //Input Patterns
	// Invoke kernel for Dot Prod
	 CudaSafeCall( hipDeviceSynchronize());
	 scalarSigProdGPU<<<128, 256>>>((float*)d_C, (int*)d_X,(int*)d_W, iTrackedCount, N);
	 CudaSafeCall( hipDeviceSynchronize());
	 CudaCheckError();
	 hipMemcpy((void*)h_sigdata,(const void*)d_C, iTrackedCount*sizeof(float), hipMemcpyDeviceToHost); //Get Result
	 
//	 for (int i=0;i<N;i++)
//		 h+=W[i]*X[i];
	 
	// h = h_odata[0]; //Test 1st Patt only
	  
	//MEasure The normalized PostSynaptic Respose As Signal
    //_Signal = h_odata[0]/_uiSynCount;
	// cout << _Signal << endl;
    
//	_SignalNTracked = hNTrack/_uiSynCount;//X[_iStoredPatIndex][_uiSynCount-1]*hNTrack/(X[0][0]*X[0][0]*(_uiSynCount-1));
	//cout << X[_iStoredPatIndex][_uiSynCount-1]/(X[0][0]*X[0][0]*(_uiSynCount-1)) << endl;
	//int iNeuronOut = ((_Signal)>0)?1:-1; //Neuron Classifier OUtput
	//cout << " Signal: " << _Signal << endl;

//	if (X[_iStoredPatIndex][_uiSynCount-1] == iNeuronOut)
//	{
//		Ret = 1; //Return 1 To indicate Successful classification of input
//	}

return iTrackedCount;
}


#endif // #ifndef _SIGNAL_KERNEL_H_
